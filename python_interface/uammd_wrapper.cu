#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2021. Doubly Periodic Stokes python bindings
   Allows to call the DPStokes module from python to compute the product between the mobility tensor and a list forces and torques acting on a group of positions.
   For additional info use:
   import uammd
   help(uammd)

*/
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include <uammd.cuh>
//Doubly Periodic FCM implementation (currently without noise)
#include <Integrator/BDHI/DoublyPeriodic/DPStokesSlab.cuh>
//Triply Periodic FCM implementation
#include <Integrator/BDHI/BDHI_FCM.cuh>

//Some convenient aliases
namespace py = pybind11;
using FCM_BM = uammd::BDHI::FCM_ns::Kernels::BarnettMagland;
using FCM = uammd::BDHI::FCM_impl<FCM_BM, FCM_BM>;
using DPStokesSlab = uammd::DPStokesSlab_ns::DPStokes;
using uammd::DPStokesSlab_ns::WallMode;
using uammd::System;
using real = uammd::real;
struct PyParameters{
  //The number of cells in each direction
  //If -1, they will be autocomputed from the tolerance if possible (DP cannot do it, FCM can)
  int nx = -1;
  int ny = -1;
  int nz = -1;
  real dt;
  real viscosity;
  real Lx;
  real Ly;
  real zmin, zmax;
  //Tolerance will be ignored in DP mode, TP will use only tolerance and nxy/nz
  real tolerance = 1e-5;
  real w, w_d;
  real hydrodynamicRadius = -1;
  real beta = -1;
  real beta_d = -1;
  real alpha = -1;
  real alpha_d = -1;
  //Can be either none, bottom, slit or periodic
  std::string mode;
};

//Helper functions and objects
struct Real3ToReal4{
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    return pr4;
  }
};
struct Real4ToReal3{
  __host__ __device__ uammd::real3 operator()(uammd::real4 i){
    auto pr3 = uammd::make_real3(i);
    return pr3;
  }
};

struct Real3ToReal4SubstractOriginZ{
  real origin;
  Real3ToReal4SubstractOriginZ(real origin):origin(origin){}
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    pr4.z -= origin;
    return pr4;
  }
};

FCM::Parameters createFCMParameters(PyParameters pypar){
  FCM::Parameters par;
  par.temperature = 0; //FCM can compute fluctuations, but they are turned off here
  par.viscosity = pypar.viscosity;
  par.tolerance = pypar.tolerance;
  par.box = uammd::Box({pypar.Lx, pypar.Ly, pypar.zmax- pypar.zmin});
  par.cells = {pypar.nx, pypar.ny, pypar.nz};
  par.kernel = std::make_shared<FCM_BM>(pypar.w, pypar.alpha, pypar.beta, pypar.Lx/pypar.nx);
  par.kernelTorque = std::make_shared<FCM_BM>(pypar.w_d, pypar.alpha_d, pypar.beta_d, pypar.Lx/pypar.nx);
  return par;
}

WallMode stringToWallMode(std::string str){
  if(str.compare("nowall") == 0){
    return WallMode::none;
  }
  else if(str.compare("slit") == 0){
    return WallMode::slit;
  }
  else if(str.compare("bottom") == 0){
    return WallMode::bottom;
  }
  else return WallMode::none;
}

DPStokesSlab::Parameters createDPStokesParameters(PyParameters pypar){
  DPStokesSlab::Parameters par;
  par.nx         = pypar.nx;
  par.ny         = pypar.ny;
  par.nz	  = pypar.nz;
  par.dt	  = pypar.dt;
  par.viscosity	  = pypar.viscosity;
  par.Lx	  = pypar.Lx;
  par.Ly	  = pypar.Ly;
  par.H		  = pypar.zmax-pypar.zmin;
  par.w = pypar.w;
  par.w_d = pypar.w_d;
  par.hydrodynamicRadius = pypar.hydrodynamicRadius;
  par.beta = pypar.beta;
  par.beta_d = pypar.beta_d;
  par.alpha = pypar.alpha;
  par.alpha_d = pypar.alpha_d;
  par.mode = stringToWallMode(pypar.mode);
  return par;
}

//Wrapper to UAMMD's TP and DP hydrodynamic modules, python interface is below
struct DPStokesUAMMD {
private:
  auto computeHydrodynamicDisplacements(bool useTorque){
    auto force = pd->getForce(uammd::access::gpu, uammd::access::read);
    auto pos = pd->getPos(uammd::access::gpu, uammd::access::read);
    auto torque = pd->getTorqueIfAllocated(uammd::access::gpu, uammd::access::read);
    auto d_torques_ptr = useTorque?torque.raw():nullptr;
    if(fcm){
      return fcm->computeHydrodynamicDisplacements(pos.raw(), force.raw(),
						   d_torques_ptr, numberParticles, st);
    }
    else if(dpstokes){
      return dpstokes->Mdot(pos.raw(), force.raw(),
			    d_torques_ptr, numberParticles, st);
    }
  }
public:
  std::shared_ptr<DPStokesSlab> dpstokes;
  std::shared_ptr<FCM> fcm;
  std::shared_ptr<uammd::System> sys;
  std::shared_ptr<uammd::ParticleData> pd;
  int numberParticles;
  hipStream_t st;
  thrust::device_vector<uammd::real3> tmp;
  real zOrigin;

  DPStokesUAMMD(PyParameters pypar, int numberParticles): numberParticles(numberParticles){
    this->sys = std::make_shared<uammd::System>();
    this->pd = std::make_shared<uammd::ParticleData>(numberParticles, sys);
    if(pypar.mode.compare("periodic")==0){
      auto par = createFCMParameters(pypar);
      this->fcm = std::make_shared<FCM>(par);
      zOrigin = 0;
    }
    else{
      auto par = createDPStokesParameters(pypar);
      this->dpstokes = std::make_shared<DPStokesSlab>(par);
      zOrigin = pypar.zmin + par.H*0.5;
    }
    CudaSafeCall(hipStreamCreate(&st));
  }

  //Copy positions to UAMMD's ParticleData
  void setPositions(py::array_t<real> h_pos){
    tmp.resize(numberParticles);
    auto pos = pd->getPos(uammd::access::gpu, uammd::access::write);
    thrust::copy((uammd::real3*)h_pos.data(), (uammd::real3*)h_pos.data() + numberParticles,
		 tmp.begin());
    thrust::transform(thrust::cuda::par.on(st), tmp.begin(), tmp.end(),
		      pos.begin(), Real3ToReal4SubstractOriginZ(zOrigin));
  }

  //Compute the hydrodynamic displacements due to a series of forces and/or torques acting on the particles
  void Mdot(py::array_t<real> h_forces, py::array_t<real> h_torques,
	    py::array_t<real> h_MF,
	    py::array_t<real> h_MT){
    // static int uses = 0;
    // uses++;
    //if(uses>=10) isNVTXEnabled = true;
    tmp.resize(numberParticles);
    bool useTorque = h_torques.size() != 0;
    {
      auto force = pd->getForce(uammd::access::gpu, uammd::access::write);
      thrust::copy((uammd::real3*)h_forces.data(), (uammd::real3*)h_forces.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par.on(st),
			tmp.begin(), tmp.end(), force.begin(), Real3ToReal4());
    }
    if(useTorque){
      auto torque = pd->getTorque(uammd::access::gpu, uammd::access::write);
      thrust::copy((uammd::real3*)h_torques.data(), (uammd::real3*)h_torques.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), torque.begin(), Real3ToReal4());
    }
    auto mob = this->computeHydrodynamicDisplacements(useTorque);
    thrust::copy(mob.first.begin(), mob.first.end(), (uammd::real3*)h_MF.mutable_data());   
    if(mob.second.size()){
      thrust::copy(mob.second.begin(), mob.second.end(), (uammd::real3*)h_MT.mutable_data());
    }    
  }
  
  ~DPStokesUAMMD(){
    hipDeviceSynchronize();
    hipStreamDestroy(st);
  }

};


//Python interface for the DPStokes module, see the accompanying example for more information
/*Usage:
  1- Call initialize with a set of parameters
  2- Call setPositions (the format must be [x0 y0 z0 x1 y1 z1,...])
  3- Call Mdot
  4- Call clear to free any memory allocated by the module and ensure a gracious finish

initialize can be called again in order to change the parameters.
Calling initialize twice is cheaper than calling initialize, then clear, then initialize again.

 */
class DPStokesPython{
  std::shared_ptr<DPStokesUAMMD> dpstokes;
public:

  //Initialize the modules with a certain set of parameters
  //Reinitializes if the module was already initialized
  void initialize(PyParameters pypar, int numberParticles){
    dpstokes = std::make_shared<DPStokesUAMMD>(pypar, numberParticles);
  }

  //Clears all memory allocated by the module.
  //This leaves the module in an unusable state until initialize is called again.
  void clear(){
    dpstokes->sys->finish();
    dpstokes.reset();
  }

  //Set positions to compute mobility matrix
  void setPositions(py::array_t<real> h_pos){
    throwIfInvalid();
    dpstokes->setPositions(h_pos);
  }

  //Compute the dot product of the mobility matrix with the forces and/or torques acting on the previously provided positions
  void Mdot(py::array_t<real> h_forces, py::array_t<real> h_torques,
	    py::array_t<real> h_MF,
	    py::array_t<real> h_MT){
    throwIfInvalid();
    dpstokes->Mdot(h_forces, h_torques, h_MF, h_MT);
  }

private:
  void throwIfInvalid(){
    if(not dpstokes){
      throw std::runtime_error("DPStokes is not initialized. Call Initialize first");
    }
  }
};

using namespace pybind11::literals;

//Transform between the enumerator for selecting a mode and a string
std::string wallModeToString(WallMode mode){
  switch(mode){
  case WallMode::none:
    return "no wall";
  case WallMode::slit:
    return "slit channel";
  case WallMode::bottom:
    return "bottom wall";
  };
}


//Pybind bindings
PYBIND11_MODULE(uammd, m) {
  m.doc() = "UAMMD DPStokes Python interface";
  py::class_<DPStokesPython>(m, "DPStokes").
    def(py::init()).
    def("initialize", &DPStokesPython::initialize,
	"Initialize the DPStokes module, can be called on an already initialize module to change the parameters.",
	"Parameters"_a, "numberParticles"_a).
    def("clear", &DPStokesPython::clear, "Release all memory allocated by the module").
    def("setPositions", &DPStokesPython::setPositions, "Set the positions to compute the mobility matrix",
	"positions"_a).
    def("Mdot", &DPStokesPython::Mdot, "Computes the product of the Mobility tensor with the provided forces and torques. If torques are not present, they are assumed to be zero and angular displacements will not be computed",
	"forces"_a, "torques"_a = py::array_t<real>(),
	"velocities"_a, "angularVelocities"_a = py::array_t<real>());
  
  py::class_<PyParameters>(m, "StokesParameters").
    def(py::init([](uammd::real viscosity,
		    uammd::real  Lx, uammd::real Ly, uammd::real zmin, uammd::real zmax,
		    uammd::real w, uammd::real w_d,
		    uammd::real alpha, uammd::real alpha_d,
		    uammd::real beta, uammd::real beta_d,
		    int Nx, int Ny, int nz, std::string mode) {
      auto tmp = std::unique_ptr<PyParameters>(new PyParameters);
      tmp->viscosity = viscosity;
      tmp->Lx = Lx;
      tmp->Ly = Ly;
      tmp->zmin = zmin;
      tmp->zmax = zmax;
      tmp->nx = Nx;
      tmp->ny = Ny;
      tmp->nz = nz;
      tmp->mode = mode;
      tmp->w = w;
      tmp->w_d = w_d;
      tmp->beta =beta;
      tmp->beta_d = beta_d;
      tmp->alpha = alpha;
      tmp->alpha_d = alpha_d;
      return tmp;	
    }),"viscosity"_a  = 1.0,"Lx"_a = 0.0, "Ly"_a = 0.0, "zmin"_a = 0.0,"zmax"_a = 0.0,
	"w"_a=1.0, "w_d"_a=1.0,
	"alpha"_a = -1.0, "alpha_d"_a=-1.0,
	"beta"_a = -1.0, "beta_d"_a=-1.0,
	"nx"_a = -1,"ny"_a = -1, "nz"_a = -1, "mode"_a="none").
    def_readwrite("viscosity", &PyParameters::viscosity, "Viscosity").
    def_readwrite("Lx", &PyParameters::Lx, "Domain size in the plane").
    def_readwrite("Ly", &PyParameters::Ly, "Domain size in the plane").
    def_readwrite("zmin", &PyParameters::zmin, "Minimum height of a particle (or bottom wall location)").
    def_readwrite("zmax", &PyParameters::zmax, "Maximum height of a particle (or top wall location)").
    def_readwrite("mode", &PyParameters::mode, "Domain walls mode, can be any of: none (no walls), bottom (wall at the bottom), slit (two walls) or periodic (uses force coupling method).").
    def_readwrite("nz", &PyParameters::nz, "Number of cells in Z").
    def_readwrite("nx", &PyParameters::nx, "Number of cells in X").
    def_readwrite("ny", &PyParameters::ny, "Number of cells in Y").
    def_readwrite("alpha", &PyParameters::alpha, "ES kernel monopole alpha").
    def_readwrite("alpha_d", &PyParameters::alpha_d, "ES kernel dipole alpha").
    def_readwrite("beta", &PyParameters::beta, "ES kernel monopole beta").
    def_readwrite("beta_d", &PyParameters::beta_d, "ES kernel dipole beta").
    def_readwrite("w", &PyParameters::w, "ES kernel monopole width").
    def_readwrite("w_d", &PyParameters::w_d, "ES kernel dipole width").
    def("__str__", [](const PyParameters &p){
      return"viscosity = " + std::to_string(p.viscosity) +"\n"+
	"box (L = " + std::to_string(p.Lx) +
	"," + std::to_string(p.Ly) + "," +
	std::to_string(p.zmin) + ":" + std::to_string(p.zmax) +" )\n"+
	"Nx = " + std::to_string(p.nx) + "\n" +
	"Ny = " + std::to_string(p.ny) + "\n" +
	"nz = " + std::to_string(p. nz) + "\n" +
	"mode = " + p.mode + "\n";
    });
    
}
