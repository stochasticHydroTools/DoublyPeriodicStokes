#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2021. Doubly Periodic Stokes python bindings
   Allows to call the DPStokes module from python to compute the product between the mobility tensor and a list forces acting on a group of Gaussian sources.
   For additional info use:
   import uammd
   help(uammd)

*/
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include <uammd.cuh>
#include <Integrator/BDHI/DoublyPeriodic/DPStokesSlab.cuh>

namespace py = pybind11;
using DPStokesSlab = uammd::DPStokesSlab_ns::DPStokes;
using Parameters = DPStokesSlab::Parameters;

struct Real3ToReal4{
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    return pr4;
  }
};
struct Real4ToReal3{
  __host__ __device__ uammd::real3 operator()(uammd::real4 i){
    auto pr3 = uammd::make_real3(i);
    return pr3;
  }
};

struct UAMMD {
  using real = uammd::real;
  std::shared_ptr<DPStokesSlab> dpstokes;
  std::shared_ptr<uammd::System> sys;
  int numberParticles;
  hipStream_t st;
  thrust::device_vector<uammd::real4> pos, force;
  thrust::device_vector<uammd::real3> tmp;
  UAMMD(Parameters par, int numberParticles): numberParticles(numberParticles){
    this->sys = std::make_shared<uammd::System>();
    this->dpstokes = std::make_shared<DPStokesSlab>(par);
    CudaSafeCall(hipStreamCreate(&st));
  }

  void Mdot(py::array_t<real> h_pos, py::array_t<real> h_forces, py::array_t<real> h_MF){
    pos.resize(numberParticles);
    force.resize(numberParticles);
    tmp.resize(numberParticles);
    thrust::copy((uammd::real3*)h_pos.data(), (uammd::real3*)h_pos.data() + numberParticles, tmp.begin());
    thrust::transform(tmp.begin(), tmp.end(), pos.begin(), Real3ToReal4());
    thrust::copy((uammd::real3*)h_forces.data(), (uammd::real3*)h_forces.data() + numberParticles, tmp.begin());
    thrust::transform(tmp.begin(), tmp.end(), force.begin(), Real3ToReal4());
    auto d_pos = thrust::raw_pointer_cast(pos.data());
    auto d_force = thrust::raw_pointer_cast(force.data());
    auto MF = dpstokes->Mdot(d_pos, d_force, numberParticles, st);
    auto MF_real3 = thrust::make_transform_iterator(MF.begin(), Real4ToReal3());
    thrust::copy(MF_real3, MF_real3 + numberParticles, (uammd::real3*)h_MF.mutable_data());
  }
  
  ~UAMMD(){
    hipDeviceSynchronize();
    hipStreamDestroy(st);
  }
};



using namespace pybind11::literals;

PYBIND11_MODULE(uammd, m) {
  m.doc() = "UAMMD DPStokes Python interface";
  py::class_<UAMMD>(m, "DPStokes").
    def(py::init<Parameters, int>(),"Parameters"_a, "numberParticles"_a).
    def("Mdot", &UAMMD::Mdot, "Computes the product of the Mobility tensor with a provided array",
	"positions"_a,"forces"_a,"result"_a);
  
  py::class_<uammd::Box>(m, "Box").
    def(py::init<uammd::real>()).
    def(py::init([](uammd::real x, uammd::real y, uammd::real z) {
      return std::unique_ptr<uammd::Box>(new uammd::Box(uammd::make_real3(x,y,z)));
    }));

  py::class_<Parameters>(m, "DPStokesParameters").
    def(py::init([](uammd::real viscosity,
		    uammd::real  Lxy, uammd::real H,
		    uammd::real gw,
		    int support, int Nxy, int nz) {             
      auto tmp = std::unique_ptr<Parameters>(new Parameters);
      tmp->viscosity = viscosity;
      tmp->box = uammd::Box(uammd::make_real3(Lxy, Lxy, H));
      tmp->box.setPeriodicity(1,1,0);
      tmp->gw = gw;
      tmp->support = support;
      tmp->cells = make_int3(Nxy, Nxy, nz);
      return tmp;	
    }),"viscosity"_a  = 1.0,"Lxy"_a = 0.0,"H"_a = 0.0,"gw"_a=1.0, "support"_a = -1, "Nxy"_a=-1, "nz"_a = -1).
    def_readwrite("viscosity", &Parameters::viscosity).
    def_readwrite("gw", &Parameters::gw).
    def_readwrite("box", &Parameters::box).
    def_readwrite("support", &Parameters::support).
    def("__str__", [](const Parameters &p){
      return"viscosity = " + std::to_string(p.viscosity) +"\n"+
	"gw = " + std::to_string(p. gw)+ "\n" +
	"box (L = " + std::to_string(p.box.boxSize.x) +
	"," + std::to_string(p.box.boxSize.y) + "," + std::to_string(p.box.boxSize.z) + ")\n"+
	"support = " + std::to_string(p. support)+ "\n" + 
	"Nxy = " + std::to_string(p. cells.x) + "\n" +
	"nz = " + std::to_string(p. cells.z) + "\n";
    });
    
}
