/* Raul P. Pelaez 2021. Doubly Periodic Stokes python bindings
   Allows to call the DPStokes module from python to compute the product between the mobility tensor and a list forces acting on a group of Gaussian sources.
   For additional info use:
   import uammd
   help(uammd)

*/
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include <uammd.cuh>
#include <Integrator/BDHI/DoublyPeriodic/DPStokesSlab.cuh>
#include <Integrator/BDHI/BDHI_FCM.cuh>


namespace py = pybind11;
using uammd::BDHI::FCM;
using DPStokesSlab = uammd::DPStokesSlab_ns::DPStokes;
using uammd::DPStokesSlab_ns::WallMode;

using real = uammd::real;
struct PyParameters{
  int nxy, nz;
  real dt;
  real viscosity;
  real Lxy;
  real H;
  real tolerance = 1e-7;
  real gw;
  int support = -1; //-1 means auto compute from tolerance
  //Can be either none, bottom, slit or periodic
  std::string mode;
};

struct Real3ToReal4{
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    return pr4;
  }
};
struct Real4ToReal3{
  __host__ __device__ uammd::real3 operator()(uammd::real4 i){
    auto pr3 = uammd::make_real3(i);
    return pr3;
  }
};

FCM::Parameters createFCMParameters(PyParameters pypar){
  FCM::Parameters par;
  par.temperature = 0;
  par.viscosity = pypar.viscosity;  
  par.tolerance = 1e-5;
  par.box = uammd::Box({pypar.Lxy, pypar.Lxy, pypar.H});
  par.cells = {pypar.nxy, pypar.nxy, pypar.nz};
  return par;
}

WallMode stringToWallMode(std::string str){
  if(str.compare("nowall") == 0){
    return WallMode::none;
  }
  else if(str.compare("slit") == 0){
    return WallMode::slit;
  }
  else if(str.compare("bottom") == 0){
    return WallMode::bottom;
  }
  else return WallMode::none;
}

DPStokesSlab::Parameters createDPStokesParameters(PyParameters pypar){
  DPStokesSlab::Parameters par;
  par.nxy         = pypar.nxy;
  par.nz	  = pypar.nz;
  par.dt	  = pypar.dt;
  par.viscosity	  = pypar.viscosity;
  par.Lxy	  = pypar.Lxy;
  par.H		  = pypar.H;
  par.gw	  = pypar.gw;
  par.support 	  = pypar.support;
  par.mode = stringToWallMode(pypar.mode);
  return par;
}

struct UAMMD {
  std::shared_ptr<DPStokesSlab> dpstokes;
  std::shared_ptr<FCM> fcm;
  std::shared_ptr<uammd::System> sys;
  std::shared_ptr<uammd::ParticleData> pd;
  int numberParticles;
  hipStream_t st;
  thrust::device_vector<uammd::real3> tmp;
  UAMMD(PyParameters pypar, int numberParticles): numberParticles(numberParticles){
    this->sys = std::make_shared<uammd::System>();
    this->pd = std::make_shared<uammd::ParticleData>(numberParticles, sys);
    if(pypar.mode.compare("periodic")==0){
      auto par = createFCMParameters(pypar);
      this->fcm = std::make_shared<FCM>(pd, sys, par);
    }
    else{
      auto par = createDPStokesParameters(pypar);
      this->dpstokes = std::make_shared<DPStokesSlab>(par);
    }
    CudaSafeCall(hipStreamCreate(&st));
  }

  void Mdot(py::array_t<real> h_pos, py::array_t<real> h_forces, py::array_t<real> h_MF){
    tmp.resize(numberParticles);
    {
      auto pos = pd->getPos(uammd::access::gpu, uammd::access::write);
      auto force = pd->getForce(uammd::access::gpu, uammd::access::write);
      std::copy((uammd::real3*)h_pos.data(), (uammd::real3*)h_pos.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), pos.begin(), Real3ToReal4());
      std::copy((uammd::real3*)h_forces.data(), (uammd::real3*)h_forces.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), force.begin(), Real3ToReal4());
    }
    auto force = pd->getForce(uammd::access::gpu, uammd::access::read);    
    if(fcm){
      auto tmp_ptr = thrust::raw_pointer_cast(tmp.data());
      fcm->Mdot(tmp_ptr, force.raw(), 0);
      thrust::copy(tmp.begin(), tmp.end(), (uammd::real3*)h_MF.mutable_data());
    }
    else if(dpstokes){
      auto pos = pd->getPos(uammd::access::gpu, uammd::access::read);
      auto MF = dpstokes->Mdot(pos.raw(), force.raw(), numberParticles, st);
      auto MF_real3 = thrust::make_transform_iterator(MF.begin(), Real4ToReal3());
      thrust::copy(MF_real3, MF_real3 + numberParticles, (uammd::real3*)h_MF.mutable_data());
    }
  }
  
  ~UAMMD(){
    hipDeviceSynchronize();
    hipStreamDestroy(st);
  }
};



using namespace pybind11::literals;


std::string wallModeToString(WallMode mode){
  switch(mode){
  case WallMode::none:
    return "no wall";
  case WallMode::slit:
    return "slit channel";
  case WallMode::bottom:
    return "bottom wall";
  };
}

PYBIND11_MODULE(uammd, m) {
  m.doc() = "UAMMD DPStokes Python interface";
  py::class_<UAMMD>(m, "DPStokes").
    def(py::init<PyParameters, int>(),"Parameters"_a, "numberParticles"_a).
    def("Mdot", &UAMMD::Mdot, "Computes the product of the Mobility tensor with a provided array",
	"positions"_a,"forces"_a,"result"_a);
  
  py::class_<PyParameters>(m, "StokesParameters").
    def(py::init([](uammd::real viscosity,
		    uammd::real  Lxy, uammd::real H,
		    uammd::real gw,
		    int support, int Nxy, int nz, std::string mode) {
      auto tmp = std::unique_ptr<PyParameters>(new PyParameters);
      tmp->viscosity = viscosity;
      tmp->Lxy = Lxy;
      tmp->H = H;      
      tmp->gw = gw;
      tmp->support = support;
      tmp->nxy = Nxy;
      tmp->nz = nz;
      tmp->mode = mode;
      return tmp;	
    }),"viscosity"_a  = 1.0,"Lxy"_a = 0.0,"H"_a = 0.0,"gw"_a=1.0, "support"_a = -1, "Nxy"_a=-1, "nz"_a = -1, "mode"_a="none").
    def_readwrite("viscosity", &PyParameters::viscosity, "Viscosity").
    def_readwrite("gw", &PyParameters::gw, "Gaussian width of the sources").
    def_readwrite("Lxy", &PyParameters::Lxy, "Domain size in the plane").
    def_readwrite("H", &PyParameters::H, "Domain width").
    def_readwrite("support", &PyParameters::support, "Number of support cells for spreading/interpolation").
    def_readwrite("mode", &PyParameters::mode, "Domain walls mode, can be any of: none (no walls), bottom (wall at the bottom), slit (two walls) or periodic (uses force coupling method).").
    def_readwrite("nz", &PyParameters::nz, "Number of cells in Z").
    def_readwrite("nxy", &PyParameters::nxy, "Number of cells in XY").
    def("__str__", [](const PyParameters &p){
      return"viscosity = " + std::to_string(p.viscosity) +"\n"+
	"gw = " + std::to_string(p. gw)+ "\n" +
	"box (L = " + std::to_string(p.Lxy) +
	"," + std::to_string(p.Lxy) + "," + std::to_string(p.H) + ")\n"+
	"support = " + std::to_string(p. support)+ "\n" + 
	"Nxy = " + std::to_string(p. nxy) + "\n" +
	"nz = " + std::to_string(p. nz) + "\n" +
	"mode = " + p.mode + "\n";
    });
    
}
