#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2021. Doubly Periodic Stokes python bindings
   Allows to call the DPStokes module from python to compute the product between the mobility tensor and a list forces and torques acting on a group of positions.
   For additional info use:
   import uammd
   help(uammd)

*/
#include "Integrator/BDHI/DoublyPeriodic/StokesSlab/utils.cuh"
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include <uammd.cuh>
#include <Integrator/BDHI/DoublyPeriodic/DPStokesSlab.cuh>
#include <Integrator/BDHI/BDHI_FCM.cuh>


namespace py = pybind11;
using uammd::BDHI::FCM;
using DPStokesSlab = uammd::DPStokesSlab_ns::DPStokes;
using uammd::DPStokesSlab_ns::WallMode;
using uammd::System;
using real = uammd::real;
struct PyParameters{
  //The number of cells in each direction
  //If -1, they will be autocomputed from the tolerance if possible (DP cannot do it, FCM can)
  int nx = -1;
  int ny = -1;
  int nz = -1;
  real dt;
  real viscosity;
  real Lx;
  real Ly;
  real zmin, zmax;
  //Tolerance will be ignored in DP mode, TP will use only tolerance and nxy/nz
  real tolerance = 1e-7;
  real w, w_d;
  real hydrodynamicRadius;
  real beta = -1;
  real beta_d = -1;
  real alpha = -1;
  real alpha_d = -1;
  //Can be either none, bottom, slit or periodic
  std::string mode;
};


struct Real3ToReal4{
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    return pr4;
  }
};
struct Real4ToReal3{
  __host__ __device__ uammd::real3 operator()(uammd::real4 i){
    auto pr3 = uammd::make_real3(i);
    return pr3;
  }
};

struct Real3ToReal4SubstractOriginZ{
  real origin;
  Real3ToReal4SubstractOriginZ(real origin):origin(origin){}
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    pr4.z -= origin;
    return pr4;
  }
};
FCM::Parameters createFCMParameters(PyParameters pypar){
  FCM::Parameters par;
  par.temperature = 0; //FCM can compute fluctuations, but they are turned off here
  par.viscosity = pypar.viscosity;
  par.tolerance = pypar.tolerance;
  par.box = uammd::Box({pypar.Lx, pypar.Ly, pypar.zmax- pypar.zmin});
  par.cells = {pypar.nx, pypar.ny, pypar.nz};
  return par;
}

WallMode stringToWallMode(std::string str){
  if(str.compare("nowall") == 0){
    return WallMode::none;
  }
  else if(str.compare("slit") == 0){
    return WallMode::slit;
  }
  else if(str.compare("bottom") == 0){
    return WallMode::bottom;
  }
  else return WallMode::none;
}

DPStokesSlab::Parameters createDPStokesParameters(PyParameters pypar){
  DPStokesSlab::Parameters par;
  par.nx         = pypar.nx;
  par.ny         = pypar.ny;
  par.nz	  = pypar.nz;
  par.dt	  = pypar.dt;
  par.viscosity	  = pypar.viscosity;
  par.Lx	  = pypar.Lx;
  par.Ly	  = pypar.Ly;
  par.H		  = pypar.zmax-pypar.zmin;
  par.w = pypar.w;
  par.w_d = pypar.w_d;
  par.hydrodynamicRadius = pypar.hydrodynamicRadius;
  par.beta = pypar.beta;
  par.beta_d = pypar.beta_d;
  par.alpha = pypar.alpha;
  par.alpha_d = pypar.alpha_d;
  par.mode = stringToWallMode(pypar.mode);
  return par;
}

struct UAMMD {
  std::shared_ptr<DPStokesSlab> dpstokes;
  std::shared_ptr<FCM> fcm;
  std::shared_ptr<uammd::System> sys;
  std::shared_ptr<uammd::ParticleData> pd;
  int numberParticles;
  hipStream_t st;
  thrust::device_vector<uammd::real3> tmp;
  real zOrigin;
  UAMMD(PyParameters pypar, int numberParticles): numberParticles(numberParticles){
    this->sys = std::make_shared<uammd::System>();
    this->pd = std::make_shared<uammd::ParticleData>(numberParticles, sys);
    if(pypar.mode.compare("periodic")==0){
      auto par = createFCMParameters(pypar);
      this->fcm = std::make_shared<FCM>(pd, sys, par);
      zOrigin = 0;
    }
    else{
      auto par = createDPStokesParameters(pypar);
      this->dpstokes = std::make_shared<DPStokesSlab>(par);
      zOrigin = pypar.zmin + par.H*0.5;
    }
    CudaSafeCall(hipStreamCreate(&st));
  }

  void Mdot(py::array_t<real> h_pos, py::array_t<real> h_forces, py::array_t<real> h_torques,
	    py::array_t<real> h_MF,
	    py::array_t<real> h_MT){
    tmp.resize(numberParticles);
    bool useTorque = h_torques.size() != 0;
    {
      auto pos = pd->getPos(uammd::access::gpu, uammd::access::write);
      auto force = pd->getForce(uammd::access::gpu, uammd::access::write);
      thrust::copy((uammd::real3*)h_pos.data(), (uammd::real3*)h_pos.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), pos.begin(), Real3ToReal4SubstractOriginZ(zOrigin));
      thrust::copy((uammd::real3*)h_forces.data(), (uammd::real3*)h_forces.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), force.begin(), Real3ToReal4());
      if(useTorque){
	auto torque = pd->getTorque(uammd::access::gpu, uammd::access::write);
        thrust::copy((uammd::real3*)h_torques.data(), (uammd::real3*)h_torques.data() + numberParticles, tmp.begin());
	thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), torque.begin(), Real3ToReal4());
      }
    }
    auto force = pd->getForce(uammd::access::gpu, uammd::access::read);
    auto pos = pd->getPos(uammd::access::gpu, uammd::access::read);
    if(fcm){
      if(h_torques.size() != 0){
    	System::log<System::EXCEPTION>("Cannot process torques in triply periodic mode");
    	throw std::runtime_error("Invalid mode");
      }
      auto tmp_ptr = thrust::raw_pointer_cast(tmp.data());
      fcm->Mdot(tmp_ptr, force.raw(), 0);
      thrust::copy(tmp.begin(), tmp.end(), (uammd::real3*)h_MF.mutable_data());
    }
    if(dpstokes){
      auto torque = pd->getTorqueIfAllocated(uammd::access::gpu, uammd::access::read);
      auto d_torques_ptr = useTorque?torque.raw():nullptr;
      //mob is a tuple containing MF and MT. The mobilities for translational and rotational contributions
      auto mob = dpstokes->Mdot(pos.raw(), force.raw(), d_torques_ptr, numberParticles, st);
      if(mob.second.size()){
	auto MT_real3 = thrust::make_transform_iterator(mob.second.begin(), Real4ToReal3());
	thrust::copy(MT_real3, MT_real3 + numberParticles, (uammd::real3*)h_MT.mutable_data());
      }
      auto MF_real3 = thrust::make_transform_iterator(mob.first.begin(), Real4ToReal3());
      thrust::copy(MF_real3, MF_real3 + numberParticles, (uammd::real3*)h_MF.mutable_data());
    }
  }
  
  ~UAMMD(){
    hipDeviceSynchronize();
    hipStreamDestroy(st);
  }
};

using namespace pybind11::literals;


std::string wallModeToString(WallMode mode){
  switch(mode){
  case WallMode::none:
    return "no wall";
  case WallMode::slit:
    return "slit channel";
  case WallMode::bottom:
    return "bottom wall";
  };
}

PYBIND11_MODULE(uammd, m) {
  m.doc() = "UAMMD DPStokes Python interface";
  py::class_<UAMMD>(m, "DPStokes").
    def(py::init<PyParameters, int>(),"Parameters"_a, "numberParticles"_a).
    def("Mdot", &UAMMD::Mdot, "Computes the product of the Mobility tensor with the provided forces and torques. If torques are not present, they are assumed to be zero and angular displacements will not be computed",
	"positions"_a,"forces"_a, "torques"_a = py::array_t<real>(),
	"velocities"_a, "angularVelocities"_a = py::array_t<real>());
  
  py::class_<PyParameters>(m, "StokesParameters").
    def(py::init([](uammd::real viscosity,
		    uammd::real  Lx, uammd::real Ly, uammd::real zmin, uammd::real zmax,
		    uammd::real w, uammd::real w_d,
		    uammd::real alpha, uammd::real alpha_d,
		    uammd::real beta, uammd::real beta_d,
		    uammd::real hydrodynamicRadius,
		    int Nx, int Ny, int nz, std::string mode) {
      auto tmp = std::unique_ptr<PyParameters>(new PyParameters);
      tmp->viscosity = viscosity;
      tmp->Lx = Lx;
      tmp->Ly = Ly;
      tmp->zmin = zmin;
      tmp->zmax = zmax;
      tmp->nx = Nx;
      tmp->ny = Ny;
      tmp->nz = nz;
      tmp->mode = mode;
      tmp->w = w;
      tmp->w_d = w_d;
      tmp->hydrodynamicRadius = hydrodynamicRadius;
      tmp->beta =beta;
      tmp->beta_d = beta_d;
      tmp->alpha = alpha;
      tmp->alpha_d = alpha_d;
      return tmp;	
    }),"viscosity"_a  = 1.0,"Lx"_a = 0.0, "Ly"_a = 0.0, "zmin"_a = 0.0,"zmax"_a = 0.0,
	"w"_a=1.0, "w_d"_a=1.0,
	"alpha"_a = -1.0, "alpha_d"_a=-1.0,
	"beta"_a = -1.0, "beta_d"_a=-1.0,
	"hydrodynamicRadius"_a = 1.0,
	"nx"_a = -1,"ny"_a = -1, "nz"_a = -1, "mode"_a="none").
    def_readwrite("viscosity", &PyParameters::viscosity, "Viscosity").
    def_readwrite("Lx", &PyParameters::Lx, "Domain size in the plane").
    def_readwrite("Ly", &PyParameters::Ly, "Domain size in the plane").
    def_readwrite("zmin", &PyParameters::zmin, "Minimum height of a particle (or bottom wall location)").
    def_readwrite("zmax", &PyParameters::zmax, "Maximum height of a particle (or top wall location)").
    def_readwrite("mode", &PyParameters::mode, "Domain walls mode, can be any of: none (no walls), bottom (wall at the bottom), slit (two walls) or periodic (uses force coupling method).").
    def_readwrite("nz", &PyParameters::nz, "Number of cells in Z").
    def_readwrite("nx", &PyParameters::nx, "Number of cells in X").
    def_readwrite("ny", &PyParameters::ny, "Number of cells in Y").
    def_readwrite("alpha", &PyParameters::alpha, "ES kernel monopole alpha").
    def_readwrite("alpha_d", &PyParameters::alpha_d, "ES kernel dipole alpha").
    def_readwrite("beta", &PyParameters::beta, "ES kernel monopole beta").
    def_readwrite("beta_d", &PyParameters::beta_d, "ES kernel dipole beta").
    def_readwrite("w", &PyParameters::w, "ES kernel monopole width").
    def_readwrite("w_d", &PyParameters::w_d, "ES kernel dipole width").
    def_readwrite("hydrodynamicRadius", &PyParameters::hydrodynamicRadius, "Hydrodynamic radius").
    def("__str__", [](const PyParameters &p){
      return"viscosity = " + std::to_string(p.viscosity) +"\n"+
	"box (L = " + std::to_string(p.Lx) +
	"," + std::to_string(p.Ly) + "," +
	std::to_string(p.zmin) + ":" + std::to_string(p.zmax) +" )\n"+
	"Nx = " + std::to_string(p.nx) + "\n" +
	"Ny = " + std::to_string(p.ny) + "\n" +
	"nz = " + std::to_string(p. nz) + "\n" +
	"mode = " + p.mode + "\n";
    });
    
}
