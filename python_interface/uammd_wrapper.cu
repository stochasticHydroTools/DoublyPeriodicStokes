#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2021. Doubly Periodic Stokes python bindings
   Allows to call the DPStokes module from python to compute the product between the mobility tensor and a list forces and torques acting on a group of positions.
   For additional info use:
   import uammd
   help(uammd)

*/
#include "Integrator/BDHI/DoublyPeriodic/StokesSlab/utils.cuh"
#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include <uammd.cuh>
#include <Integrator/BDHI/DoublyPeriodic/DPStokesSlab.cuh>
#include <Integrator/BDHI/BDHI_FCM.cuh>


namespace py = pybind11;
using uammd::BDHI::FCM;
using DPStokesSlab = uammd::DPStokesSlab_ns::DPStokes;
using uammd::DPStokesSlab_ns::WallMode;
using uammd::System;
using real = uammd::real;
struct PyParameters{
  //The number of cells in each direction
  //If -1, they will be autocomputed from the tolerance if possible (DP cannot do it, FCM can)
  int nxy = -1;
  int nz = -1;
  real dt;
  real viscosity;
  real Lxy;
  real H;
  //Tolerance will be ignored in DP mode
  real tolerance = 1e-7;
  real gw; //Gaussian width, unused with the BM kernel and in TP
  int support = -1; //-1 means auto compute from tolerance if possible (FCM can do this)
  //Can be either none, bottom, slit or periodic
  std::string mode;
};

struct Real3ToReal4{
  __host__ __device__ uammd::real4 operator()(uammd::real3 i){
    auto pr4 = uammd::make_real4(i);
    return pr4;
  }
};
struct Real4ToReal3{
  __host__ __device__ uammd::real3 operator()(uammd::real4 i){
    auto pr3 = uammd::make_real3(i);
    return pr3;
  }
};

FCM::Parameters createFCMParameters(PyParameters pypar){
  FCM::Parameters par;
  par.temperature = 0; //FCM can compute fluctuations, but they are turned off here
  par.viscosity = pypar.viscosity;
  par.tolerance = pypar.tolerance;
  par.box = uammd::Box({pypar.Lxy, pypar.Lxy, pypar.H});
  par.cells = {pypar.nxy, pypar.nxy, pypar.nz};
  return par;
}

WallMode stringToWallMode(std::string str){
  if(str.compare("nowall") == 0){
    return WallMode::none;
  }
  else if(str.compare("slit") == 0){
    return WallMode::slit;
  }
  else if(str.compare("bottom") == 0){
    return WallMode::bottom;
  }
  else return WallMode::none;
}

DPStokesSlab::Parameters createDPStokesParameters(PyParameters pypar){
  DPStokesSlab::Parameters par;
  par.nxy         = pypar.nxy;
  par.nz	  = pypar.nz;
  par.dt	  = pypar.dt;
  par.viscosity	  = pypar.viscosity;
  par.Lxy	  = pypar.Lxy;
  par.H		  = pypar.H;
  par.gw	  = pypar.gw;
  par.support 	  = pypar.support;
  par.mode = stringToWallMode(pypar.mode);
  return par;
}

struct UAMMD {
  std::shared_ptr<DPStokesSlab> dpstokes;
  std::shared_ptr<FCM> fcm;
  std::shared_ptr<uammd::System> sys;
  std::shared_ptr<uammd::ParticleData> pd;
  int numberParticles;
  hipStream_t st;
  thrust::device_vector<uammd::real3> tmp;
  UAMMD(PyParameters pypar, int numberParticles): numberParticles(numberParticles){
    this->sys = std::make_shared<uammd::System>();
    this->pd = std::make_shared<uammd::ParticleData>(numberParticles, sys);
    if(pypar.mode.compare("periodic")==0){
      auto par = createFCMParameters(pypar);
      this->fcm = std::make_shared<FCM>(pd, sys, par);
    }
    else{
      auto par = createDPStokesParameters(pypar);
      this->dpstokes = std::make_shared<DPStokesSlab>(par);
    }
    CudaSafeCall(hipStreamCreate(&st));
  }

  void Mdot(py::array_t<real> h_pos, py::array_t<real> h_forces, py::array_t<real> h_torques,
	    py::array_t<real> h_MF,
	    py::array_t<real> h_MT){
    tmp.resize(numberParticles);
    bool useTorque = h_torques.size() != 0;
    {
      auto pos = pd->getPos(uammd::access::gpu, uammd::access::write);
      auto force = pd->getForce(uammd::access::gpu, uammd::access::write);
      thrust::copy((uammd::real3*)h_pos.data(), (uammd::real3*)h_pos.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), pos.begin(), Real3ToReal4());
      thrust::copy((uammd::real3*)h_forces.data(), (uammd::real3*)h_forces.data() + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), force.begin(), Real3ToReal4());
      if(useTorque){
	auto torque = pd->getTorque(uammd::access::gpu, uammd::access::write);
        thrust::copy((uammd::real3*)h_torques.data(), (uammd::real3*)h_torques.data() + numberParticles, tmp.begin());
	thrust::transform(thrust::cuda::par, tmp.begin(), tmp.end(), torque.begin(), Real3ToReal4());
      }
    }
    auto force = pd->getForce(uammd::access::gpu, uammd::access::read);
    auto pos = pd->getPos(uammd::access::gpu, uammd::access::read);
    if(fcm){
      if(h_torques.size() != 0){
    	System::log<System::EXCEPTION>("Cannot process torques in triply periodic mode");
    	throw std::runtime_error("Invalid mode");
      }
      auto tmp_ptr = thrust::raw_pointer_cast(tmp.data());
      fcm->Mdot(tmp_ptr, force.raw(), 0);
      thrust::copy(tmp.begin(), tmp.end(), (uammd::real3*)h_MF.mutable_data());
    }
    if(dpstokes){
      auto torque = pd->getTorqueIfAllocated(uammd::access::gpu, uammd::access::read);
      auto d_torques_ptr = useTorque?torque.raw():nullptr;
      //mob is a tuple containing MF and MT. The mobilities for translational and rotational contributions
      auto mob = dpstokes->Mdot(pos.raw(), force.raw(), d_torques_ptr, numberParticles, st);
      if(mob.second.size()){
	auto MT_real3 = thrust::make_transform_iterator(mob.second.begin(), Real4ToReal3());
	thrust::copy(MT_real3, MT_real3 + numberParticles, (uammd::real3*)h_MT.mutable_data());
      }
      auto MF_real3 = thrust::make_transform_iterator(mob.first.begin(), Real4ToReal3());
      thrust::copy(MF_real3, MF_real3 + numberParticles, (uammd::real3*)h_MF.mutable_data());
    }
  }
  
  ~UAMMD(){
    hipDeviceSynchronize();
    hipStreamDestroy(st);
  }
};

using namespace pybind11::literals;


std::string wallModeToString(WallMode mode){
  switch(mode){
  case WallMode::none:
    return "no wall";
  case WallMode::slit:
    return "slit channel";
  case WallMode::bottom:
    return "bottom wall";
  };
}

PYBIND11_MODULE(uammd, m) {
  m.doc() = "UAMMD DPStokes Python interface";
  py::class_<UAMMD>(m, "DPStokes").
    def(py::init<PyParameters, int>(),"Parameters"_a, "numberParticles"_a).
    def("Mdot", &UAMMD::Mdot, "Computes the product of the Mobility tensor with the provided forces and torques. If torques are not present, they are assumed to be zero and angular displacements will not be computed",
	"positions"_a,"forces"_a, "torques"_a = py::array_t<real>(),
	"velocities"_a, "angularVelocities"_a = py::array_t<real>());
  
  py::class_<PyParameters>(m, "StokesParameters").
    def(py::init([](uammd::real viscosity,
		    uammd::real  Lxy, uammd::real H,
		    uammd::real gw,
		    int support, int Nxy, int nz, std::string mode) {
      auto tmp = std::unique_ptr<PyParameters>(new PyParameters);
      tmp->viscosity = viscosity;
      tmp->Lxy = Lxy;
      tmp->H = H;      
      tmp->gw = gw;
      tmp->support = support;
      tmp->nxy = Nxy;
      tmp->nz = nz;
      tmp->mode = mode;
      return tmp;	
    }),"viscosity"_a  = 1.0,"Lxy"_a = 0.0,"H"_a = 0.0,"gw"_a=1.0, "support"_a = -1, "Nxy"_a=-1, "nz"_a = -1, "mode"_a="none").
    def_readwrite("viscosity", &PyParameters::viscosity, "Viscosity").
    def_readwrite("gw", &PyParameters::gw, "Gaussian width of the sources").
    def_readwrite("Lxy", &PyParameters::Lxy, "Domain size in the plane").
    def_readwrite("H", &PyParameters::H, "Domain width").
    def_readwrite("support", &PyParameters::support, "Number of support cells for spreading/interpolation").
    def_readwrite("mode", &PyParameters::mode, "Domain walls mode, can be any of: none (no walls), bottom (wall at the bottom), slit (two walls) or periodic (uses force coupling method).").
    def_readwrite("nz", &PyParameters::nz, "Number of cells in Z").
    def_readwrite("nxy", &PyParameters::nxy, "Number of cells in XY").
    def("__str__", [](const PyParameters &p){
      return"viscosity = " + std::to_string(p.viscosity) +"\n"+
	"gw = " + std::to_string(p. gw)+ "\n" +
	"box (L = " + std::to_string(p.Lxy) +
	"," + std::to_string(p.Lxy) + "," + std::to_string(p.H) + ")\n"+
	"support = " + std::to_string(p. support)+ "\n" + 
	"Nxy = " + std::to_string(p. nxy) + "\n" +
	"nz = " + std::to_string(p. nz) + "\n" +
	"mode = " + p.mode + "\n";
    });
    
}
